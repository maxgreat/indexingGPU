#include "hip/hip_runtime.h"
#include "dataentry.h"

using namespace std;

float * readData(string& FileName, int size_t /* =-1 */){
	// TODO
	return nullptr;
}

DataEntry::DataEntry(size_t s){
	size = s;
	d = (float*)malloc(size*sizeof(float));

	if(d == nullptr){
		std::cerr << "Can't alloc with size" << size;
		size = 0;
	}

	srand(static_cast<unsigned int>(clock()));
	randomFill();
}

DataEntry::~DataEntry(){
	free(d);
}

bool DataEntry::changeSize(unsigned int new_size){
	size = new_size;
	d = (float*)realloc(d, new_size);
	if(d == nullptr){
		cerr << "Can't realloc with size" << new_size;
		size = 0;
		return false;
	}
	return true;
}

void DataEntry::randomFill(float minValue /*=0*/, float maxValue /*=100*/){
	for(int i=0; i < size; i++){
		d[i] = minValue + (float(rand())/float(RAND_MAX))*(maxValue-minValue);
	}
}


bool DataEntry::changeData(float* newData, unsigned int s){
	//TODO
	return false;
}


float DataEntry::distance(DataEntry& E, std::string type /*="euclidean2"*/) {
	float dist {0.0f};
	if(type.compare("euclidean2") == 0){
		for(auto i=0; i<size;i++){
			dist += (d[i]-E[i])*(d[i]-E[i]);
		}
	}
	else if(type.compare("euclidean") == 0){
		cerr << "Distance type : euclidean" << endl;
		for(auto i=0; i<size;i++){
			dist += (d[i]-E[i])*(d[i]-E[i]);
		}
		dist = sqrt(dist);
	}
	else{
		std::cerr << "Distance type unknown :" << type;
	}
	return dist;
}

void DataEntry::print(std::ostream& out) const {
	out << size << ": [";
	for (auto i = 0; i < size; i++)
		out << d[i] << " ; ";
	out << "]";
}

std::ostream& operator<< (std::ostream &out, DataEntry const& data){
	out << data.s() << ": [";
	for (auto i = 0; i < data.s()-1; i++)
		out << data[i] << ", ";
	out << data[i] <<"]";
	return out;
}




/* ************************************************************
*
*
* 	GPU Part
*
*
************************************************************ */
#ifdef with_cuda
DataEntryGPU::DataEntryGPU(unsigned int s){
	size = s;

	// Alloc Unified Memory
	auto err = hipMallocManaged((void**)&d, size*sizeof(float));


	if(err != 0){
		cout << "Error code : " << err << endl;
	}


	if(d == nullptr){
		std::cerr << "Can't alloc with size" << size << endl;
		size = 0;
	}
}


DataEntryGPU::~DataEntryGPU(){
	hipFree(d);
}


void DataEntryGPU::print(std::ostream& out) const {
	out << size << ": [";
	for (auto i = 0; i < size; i++)
		out << d[i] << " : ";
	out << "]";
}

__global__ void divideOnGPU(float* d_d, unsigned int size, float minValue /*=0*/, float maxValue){
	for(int i=0; i < 0; i++){
		d_d[i] = minValue + d_d[i]/maxValue;
	}
}

void DataEntryGPU::randomFill(float minValue /*=0*/, float maxValue /*=100*/){
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,
				HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,
				1234ULL);
	hiprandGenerateUniform(gen, d, size);
	hiprandDestroyGenerator(gen);
	divideOnGPU<<<1,1>>>(d, size, minValue, maxValue);
}

std::ostream& operator<< (std::ostream &out, DataEntryGPU const& data){
	data.print(out);
	return out;
}






#endif
