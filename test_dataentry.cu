#include "hip/hip_runtime.h"
#include "database/dataentry.h"
#include <iostream>
#include <cstdio>
#include <ctime>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

int main(){
	
	
	cout << "Unit test of DataEntry" << endl;
	DataEntry dE {10u};
	cout << "Entry of size 10: " << dE << endl;
	
	unsigned int size {200000000u};
	cout << "Test random fill on " << size*sizeof(float) << "bytes" << endl;
	DataEntry dE2 {size};
	auto c0 = clock();
	dE2.randomFill(1,10);  
	auto c1 = clock();
	cout << "randomFill in " << c1 - c0 << " clocks." << endl;

#ifdef with_cuda
	cout << "Test CUDA loading" << endl;
	DataEntryGPU d_dE {size};
	c0 = clock();
	d_dE.randomFill();  
	c1 = clock();
	cout << "randomFill in " << c1 - c0 << " clocks." << endl;
	//cout << d_dE << endl;
#endif
	return 0;

}

/*
#include <iostream>
#include <math.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged((void**)&x, N*sizeof(float));
  hipMallocManaged((void**)&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}
*/